
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>

#define CUDA_ASSERT(x) (assert((x)==hipSuccess))

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  CUDA_ASSERT(hipMalloc(&d_x, N*sizeof(float)));
  CUDA_ASSERT(hipMalloc(&d_y, N*sizeof(float)));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  CUDA_ASSERT(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
  CUDA_ASSERT(hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice));

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  CUDA_ASSERT(hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost));

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = ( maxError > abs(y[i]-4.0f) ) ? maxError : abs(y[i]-4.0f) ;
  printf("Max error: %f\n", maxError);

  CUDA_ASSERT(hipFree(d_x));
  CUDA_ASSERT(hipFree(d_y));
  free(x);
  free(y);
}
